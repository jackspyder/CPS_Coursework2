#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1 << 20;
	float *x, *y;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 1M elements on the GPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	hipEventRecord(start);
	add << <numBlocks, blockSize >> >(N, x, y);
	hipEventRecord(stop);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();


	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;
	std::cout << "Effective bandwidth(GB/s): " << N * 4 * 3 / milliseconds / 0x3E8 << std::endl;
	std::cout << N << std::endl;
	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}